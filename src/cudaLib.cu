#include "hip/hip_runtime.h"

#include "cudaLib.cuh"
#include <cstdlib>

__global__ void convLayer_gpu ( float * in, TensorShape iShape, float * filter, TensorShape fShape, float * bias, float * out, TensorShape oShape, ConvLayerArgs args, uint32_t batchSize){
	uint32_t n = blockIdx.z / oShape.channels;
	uint32_t m = blockIdx.z % oShape.channels;
	
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	

	if (x >=  oShape.height || y >= oShape.width){
		return;
	}
	
	float result = bias[m];
	for (uint32_t i = 0; i < fShape.height; ++ i) {
		for (uint32_t j = 0; j < fShape.width; ++ j) {
			for (uint32_t k = 0; k < fShape.channels; ++ k) {
				if (args.strideH*x >= iShape.height || args.strideW*y >= iShape.width){
					
				} else {
				 	result += filter[((m*fShape.channels+k)*fShape.height+i)*fShape.width+j] *in[((n*iShape.channels+k)*iShape.height+args.strideH*x)*iShape.width+args.strideW*y];				
				}	
			}
		}
	}
	out[((n*oShape.channels+m)*oShape.height+x)*oShape.width+y] = result;
	if (args.activation) {
		result = (result > 0)? result:0;
	}
}

__global__ void gemmLayer_gpu(float* A1, float* B1, float* C1, TensorShape aShape, TensorShape bShape, TensorShape cShape) {

    int x = blockIdx.x*blockDim.x+threadIdx.x;
    int y = blockIdx.y*blockDim.y+threadIdx.y;
    int n = blockIdx.z;
    float tmpSum = 0;
	float * A = &A1[n * aShape.height * aShape.width];
	float * B = &B1[n * bShape.height * bShape.width];
	float * C = &C1[n * cShape.height * cShape.width];
    if (y < cShape.width && x < cShape.height) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < aShape.width; i++) {
            tmpSum += A[x * aShape.width + i] * B[i * bShape.width + y];
        }
        C[x * cShape.width + y] = tmpSum;
    }	
}

__global__ void poolLayer_gpu (float * input, TensorShape inShape, float * output, TensorShape outShape, PoolLayerArgs args, float min_float) {
	float poolPick;
	uint32_t outRow = blockIdx.x*blockDim.x + threadIdx.x;
	uint32_t outCol = blockIdx.y*blockDim.y + threadIdx.y;
	uint32_t channel = blockIdx.z;
	if (outRow >= outShape.height || outCol >= outShape.width){
		return;
	}
	int inRow, inCol;
	poolPick = min_float;
	for (uint32_t poolRow = 0; poolRow < args.poolH; ++ poolRow) {
		for (uint32_t poolCol = 0; poolCol < args.poolW; ++ poolCol) {
			inRow = outRow *  args.strideH - (args.poolH - 1) / 2 + poolRow;
			inCol = outCol * args.strideW - (args.poolW - 1) / 2 + poolCol;
			if (inRow < 0 || inCol < 0 || inRow >= inShape.height|| inCol >= inShape.width){
			} else {
				//poolPick = (input[(inRow * inShape.width + inCol) * inShape.channels +  channel]<poolPick)?poolPick:input[(inRow * inShape.width + inCol) * inShape.channels +  channel];
				poolPick = (input[(channel*inShape.height + inCol)*inShape.width+inRow]<poolPick)?poolPick:input[(channel*inShape.height + inCol)*inShape.width+inRow];	
						
			}
		}
	}
	output[(channel*outShape.height + outCol)*outShape.width+outRow] = poolPick;
	return;
}

float * mallocTensorBatched (TensorShape shape, int batchSize) {
	if (shape.count == 0) {
		std::cout << " Shape has invalid count (4th dim) - setting to 1 \n";
		shape.count = 1;
	}
	float * d_t;
	hipMalloc((void**) &d_t, sizeof(float) * tensorSize(shape) * batchSize);
	return d_t;
}
float * makeTensorBatched (TensorShape shape, int batchSize) {
	float * t = (float *) malloc (tensorSize(shape) * sizeof(float) * batchSize);
	if (shape.count == 0) {
		std::cout << " Shape has invalid count (4th dim) - setting to 1 \n";
		shape.count = 1;
	}

	if (t == nullptr) {
		std::cout << "Malloc failed ! \n";
		return nullptr;
	}

	float * m = t;
	uint64_t offset;

	std::random_device random_device;
	std::uniform_real_distribution<float> dist(0.0, 1.0);

	//	Implement NCHW layout
	for (uint32_t count = 0; count < shape.count * batchSize; ++ count) {
		for (uint32_t chIdx = 0; chIdx < shape.channels; ++ chIdx ) {
			for (uint32_t rowIdx = 0; rowIdx < shape.height; ++ rowIdx) {
				for (uint32_t colIdx = 0; colIdx < shape.width; ++ colIdx) {
					offset = chIdx * shape.height * shape.width + rowIdx * shape.width + colIdx;
					m[offset] = dist(random_device);
				}
			}
		}
	}
	float * d_t;
	hipMalloc((void**) &d_t, sizeof(float) * tensorSize(shape) * batchSize);
	hipMemcpy(d_t, t, sizeof(float) * tensorSize(shape) * batchSize, hipMemcpyHostToDevice);
	free(t);
	return d_t;
}

float * makeVector ( uint64_t size) {
	float * v;
	v = (float *) malloc (size * sizeof(float));
	float * m = v;
	float * d_v;
	std::random_device random_device;
	std::uniform_real_distribution<float> dist(0.0, 1.0);

	//	Implement NCHW layout
	for (uint64_t idx = 0; idx < size; ++ idx) {
		m[idx] = dist(random_device);
	}
	hipMalloc((void**) &d_v, sizeof(float) * size);
	hipMemcpy(d_v, v, sizeof(float) * size, hipMemcpyHostToDevice);
	free(v);
	return d_v;
}
float * mallocVector ( uint64_t size, int batchSize) {
	float * d_v;
	hipMalloc((void**) &d_v, sizeof(float) * size * batchSize);
	return d_v;
}
void convLayer (float * d_in, TensorShape iShape, float * d_filter, TensorShape fShape, float * d_bias, float * d_out, TensorShape oShape, ConvLayerArgs args, uint32_t batchSize){
 	dim3 blockSize(16, 16);
	dim3 gridSize(oShape.height / 16 + 1, oShape.width / 16 + 1, oShape.channels *  batchSize);		
	convLayer_gpu<<<gridSize, blockSize>>>(d_in, iShape, d_filter, fShape, d_bias, d_out, oShape, args, batchSize);
	return;
}


void poolLayer (float * d_in, TensorShape iShape, float * d_out, TensorShape oShape, PoolLayerArgs_t args, uint32_t batchSize){
 	dim3 blockSize(16, 16);
	dim3 gridSize(oShape.height / 16 + 1, oShape.width / 16 + 1, oShape.channels *  batchSize);	
	TensorShape outShape = {1, iShape.channels, iShape.height / args.strideH, iShape.width / args.strideH};	
	if (outShape.height != oShape.height || outShape.width != oShape.width || outShape.channels != oShape.channels){
		std::cout << oShape << " oShape does not match! " << outShape <<std::endl;
		return;
	}
	poolLayer_gpu<<<gridSize, blockSize>>>(d_in, iShape, d_out, oShape, args, std::numeric_limits<float>::min());
	return;
}

void FCLayer (float* d_a, float* d_b, float* d_c, TensorShape aShape, TensorShape bShape, TensorShape cShape, int batchSize){
	const dim3 blockSize(16, 16);
	const dim3 gridSize(cShape.height / 16 + 1, cShape.width / 16 + 1, 1);
	aShape.height = batchSize * aShape.height;
	cShape.height = batchSize * cShape.height;
	gemmLayer_gpu<<<gridSize, blockSize>>>(d_a, d_b, d_c, aShape, bShape, cShape);

}
void AlexNet(int batchSize) {
	float ** d_filter = new float*[11];
	float ** d_in = new float*[11];
	float ** d_bias = new float*[11];
	for (int i = 0; i < 11; i++){
		d_filter[i] = nullptr;
		d_in[i] = nullptr;
		d_bias[i] = nullptr;
	}
	//init buffers
	//d_in[0] = makeTensorBatched(AlexL1_InShape, batchSize);
	d_in[1] = mallocTensorBatched(AlexL2_InShape, batchSize);
	d_in[2] = mallocTensorBatched(AlexL3_InShape, batchSize);
	d_in[3] = mallocTensorBatched(AlexL4_InShape, batchSize);
	d_in[4] = mallocTensorBatched(AlexL5_InShape, batchSize);
	d_in[5] = mallocTensorBatched(AlexL6_InShape, batchSize);
	d_in[6] = mallocTensorBatched(AlexL7_InShape, batchSize);
	d_in[7] = mallocTensorBatched(AlexL8_InShape, batchSize);
	d_in[8] = mallocTensorBatched(AlexL9_InShape, batchSize);
	d_in[9] = mallocTensorBatched(AlexL10_InShape, batchSize);
	d_in[10] = mallocTensorBatched(AlexL10_InShape, batchSize);

	
	d_filter[0] = makeTensorBatched(AlexL1_FilterShape, 1);
	d_filter[2] = makeTensorBatched(AlexL3_FilterShape, 1);
	d_filter[4] = makeTensorBatched(AlexL5_FilterShape, 1);
	d_filter[5] = makeTensorBatched(AlexL6_FilterShape, 1);
	d_filter[6] = makeTensorBatched(AlexL7_FilterShape, 1);
	d_filter[8] = makeTensorBatched(AlexL9_bShape, 1);
	d_filter[9] = makeTensorBatched(AlexL10_bShape, 1);

	d_bias[0] = makeVector(AlexL1_FilterShape.count);
	d_bias[2] = makeVector(AlexL3_FilterShape.count);
	d_bias[4] = makeVector(AlexL5_FilterShape.count);
	d_bias[5] = makeVector(AlexL6_FilterShape.count);
	d_bias[6] = makeVector(AlexL7_FilterShape.count);
	//input 


	//layers
	//layer1
	for (int i = 0; i < 20;i++){
		std::cout << "batch" << i << std::endl;
		d_in[0] = makeTensorBatched(AlexL1_InShape, batchSize);
		convLayer(d_in[0], AlexL1_InShape, d_filter[0], AlexL1_FilterShape, d_bias[0], d_in[1], AlexL2_InShape, AlexL1_ConvArgs, batchSize);
		poolLayer(d_in[1], AlexL2_InShape, d_in[2], AlexL3_InShape, AlexL2_PoolArgs, batchSize);
		convLayer(d_in[2], AlexL3_InShape, d_filter[2], AlexL3_FilterShape, d_bias[2], d_in[3], AlexL4_InShape, AlexL3_ConvArgs, batchSize);
		poolLayer(d_in[3], AlexL4_InShape, d_in[4], AlexL5_InShape, AlexL4_PoolArgs, batchSize);
		convLayer(d_in[4], AlexL5_InShape, d_filter[4], AlexL5_FilterShape, d_bias[4], d_in[5], AlexL6_InShape, AlexL5_ConvArgs, batchSize);
		convLayer(d_in[5], AlexL6_InShape, d_filter[5], AlexL6_FilterShape, d_bias[5], d_in[6], AlexL7_InShape, AlexL6_ConvArgs, batchSize);
		convLayer(d_in[6], AlexL7_InShape, d_filter[6], AlexL7_FilterShape, d_bias[6], d_in[7], AlexL8_InShape, AlexL7_ConvArgs, batchSize);
		poolLayer(d_in[7], AlexL8_InShape, d_in[8], AlexL9_InShape, AlexL8_PoolArgs, batchSize);

		FCLayer(d_in[8], d_filter[8], d_in[9], AlexL9_aShape, AlexL9_bShape, AlexL9_cShape, batchSize);
		FCLayer(d_in[9], d_filter[9], d_in[10], AlexL10_aShape, AlexL10_bShape, AlexL10_cShape, batchSize);
	}
	//free Memory
	for (int i = 0; i < 11; i++){
		if (d_filter[i] != nullptr){		
			hipFree(d_filter[i]); 
		}
		if (d_in[i] != nullptr){
			hipFree(d_in[i]); 
		}
		if (d_bias[i] != nullptr){
			hipFree(d_bias[i]); 
		}
	}
	return;
}

